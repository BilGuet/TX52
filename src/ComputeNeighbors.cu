#include "hip/hip_runtime.h"
#include <iostream>

#include "Point.h"

__device__ double get_maximum(double* distance, int& index)
{
    double max = 0;

    for(int i = 0; i < k; i++)
    {
        if(distance[i] > max)
        {
            max = distance[i];
            index = i;
        }
    }

    return max;
}

__global__ void ComputeNeighbors(Point* points, size_t* AllNeighbors, size_t n)
{
    // this thread calculate for points associates to it by id
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    // gridDim.x*blockDim.x = number of threads
    for(int p = id; p < n; p += gridDim.x*blockDim.x)
    {
        double distance [k];
        for (int i = 0; i < k; i++)
        {
            distance[i] = 100000;
        }

        for (size_t q = 0; q < n; q ++)
        {
            //check that we're not calculating the distance between p and itself
            if (q != p)
            {
                // calcuate the distance between p and q
                double d = sqrt(pow(points[p].x - points[q].x, 2) + pow(points[p].y - points[q].y, 2));

                int index;
                double max = get_maximum(distance, index);

                if(d < max)
                {
                    distance[index] = d;
                    AllNeighbors[p*k + index] = q;
                }
            }
        }
    }
}
