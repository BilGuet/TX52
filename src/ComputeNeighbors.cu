#include "hip/hip_runtime.h"
#include <iostream>

#include "Point.h"

__device__ double get_maximum(double* distance, int& index, int p, unsigned int k)
{
    double max = 0;

    for(int i = 0; i < k; i++)
    {
        if(distance[p*k + i] > max)
        {
            max = distance[p*k + i];
            index = i;
        }
    }

    return max;
}

__global__ void ComputeNeighbors(Point* points, size_t* AllNeighbors, double* distance, size_t n, unsigned int k)
{
    // this thread calculate for points associates to it by id
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    // gridDim.x*blockDim.x = number of threads
    for(int p = id; p < n; p += gridDim.x*blockDim.x)
    {
        //double distance [k];
        for (int i = 0; i < k; i++)
        {
            distance[p*k + i] = 100000;
        }

        for (size_t q = 0; q < n; q ++)
        {
            //check that we're not calculating the distance between p and itself
            if (q != p)
            {
                // calcuate the distance between p and q
                double d = sqrt(pow(points[p].x - points[q].x, 2) + pow(points[p].y - points[q].y, 2));

                int index;
                double max = get_maximum(distance, index, p, k);

                //if the distance is lower than the biggest distance, insert that point in neighbors
                if(d < max)
                {
                    distance[p*k + index] = d;
                    AllNeighbors[p*k + index] = q;
                }
            }
        }
    }
}
