#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cassert>
#include "KNearestNeighbors.cuh"
#include "ComputeNeighbors.cuh"

////    CPU Version of Neighbors algorithm  ////
void GetKNearestNeighborsCPU(const size_t p, const std::vector<Point>& points, std::vector<size_t>& neighbors, unsigned int k)
{
    neighbors.resize(k);
    std::vector<double> distance(k, 1000);

    for (size_t q = 0; q < points.size(); q++)
    {
        //check that we're not calculating the distance between p and itself
        if (q != p)
        {
            //calcuate the distance between p and q
            double d = sqrt(pow(points[p].x - points[q].x, 2) + pow(points[p].y - points[q].y, 2));

            //check if q is nearer than the farest of the nearest point
            auto max = std::max_element(distance.begin(), distance.end());
            if (d < *max)
            {
                // store the distance and index of q
                distance[std::distance(distance.begin(), max)] = d;
                neighbors[std::distance(distance.begin(), max)] = q;
            }
        }
    }
}


void GetKNearestNeighborsGPU(const std::vector<Point>& points, std::vector< std::vector<size_t> >& AllNeighbors, unsigned int k)
{
    std::vector<size_t> neighbors;
    
    Point* CPUpoints = (Point*)malloc(points.size() * sizeof(Point));
    size_t* CPUneighbors = (size_t*)malloc(points.size() * k * sizeof(size_t));
    
    // instanciate points coordinates
    for(int i = 0; i < points.size(); i++)
    {
        CPUpoints[i] = points[i];
    }


    // GPU variables
    Point* GPUpoints;
    size_t* GPUneighbors;
    double* GPUdistance;
    assert(hipMalloc((void**)&GPUpoints, points.size() * sizeof(Point)) == hipSuccess);
    assert(hipMalloc((void**)&GPUneighbors, points.size() * k * sizeof(size_t)) == hipSuccess);
    assert(hipMalloc((void**)&GPUdistance, points.size()*k * sizeof(double)) == hipSuccess);


    // send points coordinates to GPU memory
    assert(hipMemcpy(GPUpoints, CPUpoints, points.size() * sizeof(Point), hipMemcpyHostToDevice) == hipSuccess);
    
    std::cout << "Computing neighbors..." << std::endl;
    ComputeNeighbors<<< (points.size()/512)+1, 512 >>>(GPUpoints, GPUneighbors, GPUdistance, points.size(), k);
    hipDeviceSynchronize();
    
    // recover the neighbors indexes from GPU memory
    assert(hipMemcpy(CPUneighbors, GPUneighbors, points.size() * k * sizeof(size_t), hipMemcpyDeviceToHost) == hipSuccess);

    // make sure that neighbors vector is at good size
    neighbors.resize(k);
    // make sure that AllNeighbors vector is empty
    AllNeighbors.clear();

    for(int i = 0; i < points.size(); i++)
    {
        for(int j = 0; j < k; j++)
        {
            neighbors[j] = CPUneighbors[i*k + j];
        }

        // ad vector of neighbors to vector of all neighbors
        AllNeighbors.push_back(neighbors);
    }


    free(CPUpoints);
    free(CPUneighbors);
    hipFree(GPUpoints);
    hipFree(GPUneighbors);
    hipFree(GPUdistance);
}
